
#include <cublas_matrix_multiplier.hpp>
#include <hip/hip_runtime.h>
#include <cassert>
#include <hipblas.h>
#include <cstdio>

template <long N>
CublasMatrixMultiplier<N>::CublasMatrixMultiplier()
{
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
}

template <long N>
CublasMatrixMultiplier<N>::~CublasMatrixMultiplier()
{
    hipblasDestroy(cublasHandle);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

template <long N>
void CublasMatrixMultiplier<N>::multiply(float (&A)[N][N], float (&B)[N][N], float (&C)[N][N])
{
    hipError_t cudaStatus = hipMalloc(&A_d, N * N * sizeof(*A_d));
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMalloc(&B_d, N * N * sizeof(*B_d));
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMalloc(&C_d, N * N * sizeof(*C_d));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy2D(A_d, N * sizeof(float), A, N * sizeof(float), N * sizeof(float), N, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy2D(B_d, N * sizeof(float), B, N * sizeof(float), N * sizeof(float), N, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);

    const float alpha = 1;
    const float beta = 0;

    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, B_d, N, A_d, N, &beta, C_d, N);

    cudaStatus = hipMemcpy2D(C, N * sizeof(float), C_d, N * sizeof(float), N * sizeof(float), N, hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);
}

// explicit instantion: https://docs.microsoft.com/en-us/cpp/cpp/explicit-instantiation?view=msvc-170
template class CublasMatrixMultiplier<1024>;


#include <cublas_matrix_multiplier.hpp>
#include <hip/hip_runtime.h>
#include <cassert>
#include <hipblas.h>
#include <cstdio>

template <long N>
CublasMatrixMultiplier<N>::CublasMatrixMultiplier()
{
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
}

template <long N>
CublasMatrixMultiplier<N>::~CublasMatrixMultiplier()
{
    hipblasDestroy(cublasHandle);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

template <long N>
void CublasMatrixMultiplier<N>::multiply(float (&A)[N][N], float (&B)[N][N], float (&C)[N][N])
{
    hipError_t cudaStatus = hipMalloc(&A_d, N * N * sizeof(*A_d));
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMalloc(&B_d, N * N * sizeof(*B_d));
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMalloc(&C_d, N * N * sizeof(*C_d));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy2D(A_d, 0, A, 0, N, N, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy2D(B_d, 0, B, 0, N, N, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);

    const float alpha = 1;
    const float beta = 0;

    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, A_d, N, B_d, N, &beta, C_d, N);

    hipMemcpy2D(C, 0, C_d, 0, N, N, hipMemcpyDeviceToHost);
}

// explicit instantion: https://docs.microsoft.com/en-us/cpp/cpp/explicit-instantiation?view=msvc-170
template class CublasMatrixMultiplier<1024>;
